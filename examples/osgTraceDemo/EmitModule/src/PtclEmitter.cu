
#include <hip/hip_runtime.h>
/* osgCompute - Copyright (C) 2008-2009 SVT Group
*                                                                     
* This library is free software; you can redistribute it and/or modify
* it under the terms of the GNU Lesser General Public License as
* published by the Free Software Foundation; either version 3 of
* the License, or (at your option) any later version.
*                                                                     
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of 
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU Lesse General Public License for more details.
*
* The full license is in LICENSE file included with this distribution.
*/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float lerp(float a, float b, float t)
{
    return a + t*(b-a);
}

//------------------------------------------------------------------------------
inline __device__ 
float4 reseed( float* seeds, unsigned int seedCount, unsigned int seedIdx, unsigned int ptclIdx, float3 bbmin, float3 bbmax )
{
    // random seed idx
    unsigned int idx1 = (seedIdx + ptclIdx) % seedCount;
    unsigned int idx2 = (idx1 + ptclIdx) % seedCount;
    unsigned int idx3 = (idx2 + ptclIdx) % seedCount;

    // seeds are within the range [0,1]
    float intFac1 = seeds[idx1];
    float intFac2 = seeds[idx2];
    float intFac3 = seeds[idx3];

    return make_float4(lerp(bbmin.x,bbmax.x,intFac1), lerp(bbmin.y,bbmax.y,intFac3),
        lerp(bbmin.z,bbmax.z,intFac2), 1);
}

//------------------------------------------------------------------------------
inline __device__
unsigned int thIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void emitKernel(  unsigned int numPtcls, float4* ptcls, float* seeds, unsigned int seedIdx, float3 bbmin, float3 bbmax )
{
    unsigned int ptclIdx = thIdx();
    if( ptclIdx < numPtcls )
    {
        float4 curPtcl = ptcls[ptclIdx];

        if( curPtcl.x < bbmin.x ||
            curPtcl.y < bbmin.y ||
            curPtcl.z < bbmin.z ||
            curPtcl.x > bbmax.x ||
            curPtcl.y > bbmax.y ||
            curPtcl.z > bbmax.z )
            ptcls[ptclIdx] = reseed( seeds, numPtcls, seedIdx, ptclIdx, bbmin, bbmax );
    }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
extern "C" __host__
void emit(unsigned int numPtcls, 
            void* ptcls, 
            void* seeds, 
            unsigned int seedIdx, 
            float3 bbmin, 
            float3 bbmax )
{
    dim3 blocks( (numPtcls / 128)+1, 1, 1 );
    dim3 threads( 128, 1, 1 );

    emitKernel<<< blocks, threads >>>(
        numPtcls,
        reinterpret_cast<float4*>(ptcls),
        reinterpret_cast<float*>(seeds),
        seedIdx,
        bbmin,
        bbmax);
}
